#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "libcuda.h"


__global__ void kernel_VectorAddScalar(float *v, float *scalar)
{
    int thread_idx = threadIdx.x + blockIdx.x*blockDim.x;
    v[thread_idx] = v[thread_idx] + *scalar;
}

int CcuVectorAddScalar::Add(float *v, float scalar)
{
   hipError_t error_code = hipSuccess;

    // allocate v's replica in device mem
    float *d_v = NULL;
    error_code = hipMalloc((void **)&d_v, 1e6*sizeof(float));
    if (error_code != hipSuccess)
        return error_code;

    // allocate scalar replica in device mem
    float *d_scalar = NULL;
    error_code = hipMalloc((void **)&d_scalar, sizeof(float));
    if (error_code != hipSuccess)
        return error_code;

    // copy vector to device mem
    error_code = hipMemcpy(d_v, v, 1e6*sizeof(float), hipMemcpyHostToDevice);
    if (error_code != hipSuccess)
        return error_code;
    // copy scalar to device mem
    error_code = hipMemcpy(d_scalar, &scalar, sizeof(float), hipMemcpyHostToDevice);
    if (error_code != hipSuccess)
        return error_code;

    // invoke kernel
    kernel_VectorAddScalar<<<1000, 1000>>>(d_v, d_scalar);
    error_code = hipGetLastError();
    if (error_code != hipSuccess)
        return error_code;

    // copy result back to host (in place)
    error_code = hipMemcpy(v, d_v, 1e6*sizeof(float), hipMemcpyDeviceToHost);
    if (error_code != hipSuccess)
        return error_code;

    return 0;

 }